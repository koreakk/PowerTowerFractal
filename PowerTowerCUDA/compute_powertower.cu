#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/complex.h>

#include <opencv2/opencv.hpp>

//Define a CUDA kernel function to compute the power tower fractal
__global__ void compute_powertower_kernel(int rows, int cols, float range_left, float range_right, float range_top, float range_bottom, 
                                          int max_iterations, int threshold, uchar* __restrict__ data)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (y >= rows || x >= cols) { return; }

    float scaleX = cols / (range_right - range_left);
    float scaleY = rows / (range_bottom - range_top);

    float real = x / scaleX + range_left;
    float imag = y / scaleY + range_top;
    thrust::complex<float> c(real, imag);
    thrust::complex<float> z(c);

    for (int i = 0; i < max_iterations; ++i) {
        z = thrust::pow(c, z);
        if (thrust::abs(z) > threshold) {
            data[y * cols + x] = 255;
            return;
        }
    }
    data[y * cols + x] = 0;
}

cv::Mat compute_powertower(int rows, int cols, float range_left, float range_right, float range_top, float range_bottom,
                           int max_iterations, int threshold)
{
    // Allocate memory on the GPU
    uchar* data;
    hipMalloc((void**)&data, (size_t)rows * cols * sizeof(uchar));

    // Execute the CUDA kernel on the GPU
    dim3 blockSize(32, 32);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y);
    compute_powertower_kernel <<< gridSize, blockSize >>> (rows, cols, range_left, range_right, range_top, range_bottom,
                                                            max_iterations, threshold, data);

    // Check for any errors during kernel execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }

    // Copy the result from the GPU to the host
    cv::Mat powertower_set(rows, cols, CV_8UC1);
    hipMemcpy(powertower_set.data, data, (size_t)rows * cols * sizeof(uchar), hipMemcpyDeviceToHost);

    // Check for any errors during memory copy
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }

    // Free the GPU memory
    hipFree(data);

    return powertower_set;
}